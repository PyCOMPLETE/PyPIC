/*
   GPU Kernels for the particles to mesh functions
   @author: Stefan Hegglin, Adrian Oeftiger
*/

#include <hip/hip_runtime.h>

// implementation from: http://docs.nvidia.com/cuda/cuda-c-programming-guide/#atomicadd
// very slow, for <NVIDIA P100 purposes where double atomicAdd does not exist yet
// 2017-05-22 edit based on:
// http://stackoverflow.com/questions/39274472/error-function-atomicadddouble-double-has-already-been-defined

/*
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 600

#else
static __inline__ __device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    if (val==0.0)
        return __longlong_as_double(old);
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif
*/

#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ < 600
static __inline__ __device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    if (val==0.0)
        return __longlong_as_double(old);
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif
#endif


extern "C" {

__global__ void particles_to_mesh_2d_64atomics(
        int nparticles, double *grid1d, const int nx, const int ny,
        double *wij, double *wi1j, double *wij1, double *wi1j1, int *i, int *j)

{
    int pidx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y* blockDim.x + threadIdx.x;
    int ix = i[pidx];
    int jx = j[pidx];
    if (pidx < nparticles) {
        if (jx >= 0 && jx < nx - 1 && ix >= 0 && ix < ny - 1)
        {
            atomicAdd(&grid1d[jx + ix*nx], wij[pidx]);
            atomicAdd(&grid1d[jx+1 + ix*nx], wij1[pidx]);
            atomicAdd(&grid1d[jx + (ix+1)*nx], wi1j[pidx]);
            atomicAdd(&grid1d[jx+1 + (ix+1)*nx], wi1j1[pidx]);
        }
    }
}

__global__ void particles_to_mesh_3d_64atomics(
        int nparticles,
        double *grid1d, const int nx, const int ny, const int nz,
        // particle weights:
        double *wijk, double *wi1jk, double *wij1k, double *wi1j1k,
        double *wijk1, double *wi1jk1, double* wij1k1, double* wi1j1k1,
        // particle 3d cell indices
        int *i, int *j, int* k)
{
    int pidx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y* blockDim.x + threadIdx.x;
    int ix = i[pidx];
    int jx = j[pidx];
    int kx = k[pidx];
    if (pidx < nparticles) {
        if (jx >= 0 && jx < nx - 1 && ix >= 0 && ix < ny - 1 && kx >= 0 && kx < nz - 1)
        {
            atomicAdd(&grid1d[jx   + ix*nx     + kx*nx*ny],     wijk[pidx]);
            atomicAdd(&grid1d[jx+1 + ix*nx     + kx*nx*ny],     wij1k[pidx]);
            atomicAdd(&grid1d[jx   + (ix+1)*nx + kx*nx*ny],     wi1jk[pidx]);
            atomicAdd(&grid1d[jx+1 + (ix+1)*nx + kx*nx*ny],     wi1j1k[pidx]);
            atomicAdd(&grid1d[jx   + ix*nx     + (kx+1)*nx*ny], wijk1[pidx]);
            atomicAdd(&grid1d[jx+1 + ix*nx     + (kx+1)*nx*ny], wij1k1[pidx]);
            atomicAdd(&grid1d[jx   + (ix+1)*nx + (kx+1)*nx*ny], wi1jk1[pidx]);
            atomicAdd(&grid1d[jx+1 + (ix+1)*nx + (kx+1)*nx*ny], wi1j1k1[pidx]);
        }
    }
}

__global__ void particles_to_mesh_2d(
        int nparticles, float *grid1d, const int nx, const int ny,
        float *wij, float *wi1j, float *wij1, float *wi1j1, int *i, int *j)

{
    int pidx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y* blockDim.x + threadIdx.x;
    int ix = i[pidx];
    int jx = j[pidx];
    if (pidx < nparticles) {
        if (jx >= 0 && jx < nx - 1 && ix >= 0 && ix < ny - 1)
        {
            atomicAdd(&grid1d[jx   + ix*nx],     wij[pidx]);
            atomicAdd(&grid1d[jx+1 + ix*nx],     wij1[pidx]);
            atomicAdd(&grid1d[jx   + (ix+1)*nx], wi1j[pidx]);
            atomicAdd(&grid1d[jx+1 + (ix+1)*nx], wi1j1[pidx]);
        }
    }
}

__global__ void particles_to_mesh_3d(
        int nparticles,
        float *grid1d, const int nx, const int ny, const int nz,
        // particle weights:
        float *wijk, float *wi1jk, float *wij1k, float *wi1j1k,
        float *wijk1, float *wi1jk1, float* wij1k1, float* wi1j1k1,
        // particle 3d cell indices
        int *i, int *j, int* k)
{
    int pidx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y* blockDim.x + threadIdx.x;
    int ix = i[pidx];
    int jx = j[pidx];
    int kx = k[pidx];
    if (pidx < nparticles) {
        if (jx >= 0 && jx < nx - 1 && ix >= 0 && ix < ny - 1 && kx >= 0 && kx < nz - 1)
        {
            atomicAdd(&grid1d[jx   + ix*nx     + kx*nx*ny],     wijk[pidx]);
            atomicAdd(&grid1d[jx+1 + ix*nx     + kx*nx*ny],     wij1k[pidx]);
            atomicAdd(&grid1d[jx   + (ix+1)*nx + kx*nx*ny],     wi1jk[pidx]);
            atomicAdd(&grid1d[jx+1 + (ix+1)*nx + kx*nx*ny],     wi1j1k[pidx]);
            atomicAdd(&grid1d[jx   + ix*nx     + (kx+1)*nx*ny], wijk1[pidx]);
            atomicAdd(&grid1d[jx+1 + ix*nx     + (kx+1)*nx*ny], wij1k1[pidx]);
            atomicAdd(&grid1d[jx   + (ix+1)*nx + (kx+1)*nx*ny], wi1jk1[pidx]);
            atomicAdd(&grid1d[jx+1 + (ix+1)*nx + (kx+1)*nx*ny], wi1j1k1[pidx]);
        }
    }
}

__global__ void cic_guard_cell_weights_3d(
        // particle positions sorted by cell ids
        double *x_sorted, double *y_sorted, double *z_sorted,
        // mesh
        double x0, double y0, double z0,
        double dx, double dy, double dz,
        int nx, int ny, int n_nodes,
        int* lower_bounds, int* upper_bounds,
        // output: cumulative mesh charges for guard cells
        double* cumweight_ijk, double* cumweight_i1jk,
        double* cumweight_ij1k, double* cumweight_i1j1k,
        double* cumweight_ijk1, double* cumweight_i1jk1,
        double* cumweight_ij1k1, double* cumweight_i1j1k1)
/**
    Calculate the Cloud-in-Cell weights for all particles within a
    guard cell.

    This node-based algorithm expects particle arrays sorted
    by their node id. For each node, the corresponding guard cell
    is spanned from the nodes spatial indices i, j, k to i+1, j+1, k+1.
    (Therefore, the guard cells at the rear boundary nodes do not
    get any contribution for the guard cell nodes that lie outside of
    the original mesh. E.g. i+1 == ny gets a zero entry for sure!)
    Within a guard cell, all particles are weighted according to
    their distance to the respective bounding node.

    The weights of each particle are summed up per guard cell node and
    written back to the global arrays cumweight_ijk etc.

    The index arrays lower_bounds and upper_bounds
    indicate the start and end indices
    within the sorted particle arrays for each node id. The respective
    node id is identical to the index within lower_bounds and
    upper_bounds.
*/
{
    double l_cumweight_ijk,  l_cumweight_i1jk,  l_cumweight_ij1k,  l_cumweight_i1j1k,
           l_cumweight_ijk1, l_cumweight_i1jk1, l_cumweight_ij1k1, l_cumweight_i1j1k1;
    int i, j, k;
    double x0bydx = x0/dx;
    double y0bydy = y0/dy;
    double z0bydz = z0/dz;
    double dx_rel, dy_rel, dz_rel;
    // grid-stride loop
    for (int nid = blockIdx.x * blockDim.x + threadIdx.x;
         nid < n_nodes;
         nid += blockDim.x * gridDim.x)
    {
        j = nid % nx; //& (nx-1); //
        i = ((nid - j) / nx) % ny; //& (ny-1); //
        k = (nid - j - nx * i) / (nx * ny);

        l_cumweight_ijk = 0.;   l_cumweight_i1jk = 0.; l_cumweight_ij1k = 0.;
        l_cumweight_i1j1k = 0.; l_cumweight_ijk1 = 0.; l_cumweight_i1jk1 = 0.;
        l_cumweight_ij1k1 = 0.; l_cumweight_i1j1k1 = 0.;
        for (int pid = lower_bounds[nid]; pid < upper_bounds[nid]; pid++)
        {
            dx_rel = x_sorted[pid]/dx - x0bydx - j;
            dy_rel = y_sorted[pid]/dy - y0bydy - i;
            dz_rel = z_sorted[pid]/dz - z0bydz - k;

            // locally calculate the weights for all 8 nodes of current guard cell
            l_cumweight_ijk +=    (1-dx_rel)*(1-dy_rel)*(1-dz_rel);
            l_cumweight_i1jk +=   (1-dx_rel)*(dy_rel)  *(1-dz_rel);
            l_cumweight_ij1k +=   (dx_rel)  *(1-dy_rel)*(1-dz_rel);
            l_cumweight_i1j1k +=  (dx_rel)  *(dy_rel)  *(1-dz_rel);
            l_cumweight_ijk1 +=   (1-dx_rel)*(1-dy_rel)*(dz_rel);
            l_cumweight_i1jk1 +=  (1-dx_rel)*(dy_rel)  *(dz_rel);
            l_cumweight_ij1k1 +=  (dx_rel)  *(1-dy_rel)*(dz_rel);
            l_cumweight_i1j1k1 += (dx_rel)  *(dy_rel)  *(dz_rel);
        }
        cumweight_ijk[nid] =   l_cumweight_ijk;   cumweight_i1jk[nid] =   l_cumweight_i1jk;
        cumweight_ij1k[nid] =  l_cumweight_ij1k;  cumweight_i1j1k[nid] =  l_cumweight_i1j1k;
        cumweight_ijk1[nid] =  l_cumweight_ijk1;  cumweight_i1jk1[nid] =  l_cumweight_i1jk1;
        cumweight_ij1k1[nid] = l_cumweight_ij1k1; cumweight_i1j1k1[nid] = l_cumweight_i1j1k1;
    }
}

__global__ void join_guard_cells_3d_old( // excl boundary
        double* cumweight_ijk, double* cumweight_i1jk,
        double* cumweight_ij1k, double* cumweight_i1j1k,
        double* cumweight_ijk1, double* cumweight_i1jk1,
        double* cumweight_ij1k1, double* cumweight_i1j1k1,
        int n_nodes, int nx, int ny, int nz,
        double* mesh_charges)
/**

*/
{
    int i, j, k, ijk, i1jk, ij1k, i1j1k, ijk1, i1jk1, ij1k1, i1j1k1;
    // grid-stride loop
    for (int nid = blockIdx.x * blockDim.x + threadIdx.x;
         nid < n_nodes;
         nid += blockDim.x * gridDim.x)
    {
        j = nid % nx; //& (nx-1); //
        i = ((nid - j) / nx) % ny; //& (ny-1); //
        k = (nid - j - nx * i) / (nx * ny);
        if (j == 0 || j == nx - 1 || i == 0 || i == ny - 1 || k == 0 || k == nz - 1)
        {
            continue;
        }

        ijk = nid;                     // nx*ny*k     + nx*i     + j
        i1jk = nid - nx;               // nx*ny*k     + nx*(i-1) + j
        ij1k = nid - 1;                // nx*ny*k     + nx*i     + j-1
        i1j1k = nid - nx - 1;          // nx*ny*k     + nx*(i-1) + j-1
        ijk1 = nid - nx*ny;            // nx*ny*(k-1) + nx*i     + j
        i1jk1 = nid - nx*ny - nx;      // nx*ny*(k-1) + nx*(i-1) + j
        ij1k1 = nid - nx*ny - 1;       // nx*ny*(k-1) + nx*i     + j-1
        i1j1k1 = nid - nx*ny - nx - 1; // nx*ny*(k-1) + nx*(i-1) + j-1


        mesh_charges[nid] =   cumweight_ijk[ijk]     + cumweight_i1jk[i1jk]
                            + cumweight_ij1k[ij1k]   + cumweight_i1j1k[i1j1k]
                            + cumweight_ijk1[ijk1]   + cumweight_i1jk1[i1jk1]
                            + cumweight_ij1k1[ij1k1] + cumweight_i1j1k1[i1j1k1];

    }
}

__global__ void join_guard_cells_3d( // _incl_boundary
        double* cumweight_ijk, double* cumweight_i1jk,
        double* cumweight_ij1k, double* cumweight_i1j1k,
        double* cumweight_ijk1, double* cumweight_i1jk1,
        double* cumweight_ij1k1, double* cumweight_i1j1k1,
        int n_nodes, int nx, int ny, int nz,
        double* mesh_charges)
/**

*/
{
    double l_mesh_charges;
    int j, i, k;
    // grid-stride loop
    for (int nid = blockIdx.x * blockDim.x + threadIdx.x;
         nid < n_nodes;
         nid += blockDim.x * gridDim.x)
    {
        j = nid % nx; //& (nx-1); //
        i = ((nid - j) / nx) % ny; //& (ny-1); //
        k = (nid - j - nx * i) / (nx * ny);

        l_mesh_charges = cumweight_ijk[nid];
        if (j > 0)
            l_mesh_charges += cumweight_ij1k[nid - 1];
        if (i > 0)
            l_mesh_charges += cumweight_i1jk[nid - nx];
        if (j > 0 && i > 0)
            l_mesh_charges += cumweight_i1j1k[nid - nx - 1];
        if (k > 0)
            l_mesh_charges += cumweight_ijk1[nid - nx*ny];
        if (j > 0 && k > 0)
            l_mesh_charges += cumweight_ij1k1[nid - nx*ny - 1];
        if (i > 0 && k > 0)
            l_mesh_charges += cumweight_i1jk1[nid - nx*ny - nx];
        if (j > 0 && i > 0 && k > 0)
            l_mesh_charges += cumweight_i1j1k1[nid - nx*ny - nx - 1];

        mesh_charges[nid] = l_mesh_charges;
    }
}


__global__ void cic_guard_cell_weights_2d(
        // particle positions sorted by cell ids
        double *x_sorted, double *y_sorted,
        // mesh
        double x0, double y0,
        double dx, double dy,
        int nx, int n_nodes,
        int* lower_bounds, int* upper_bounds,
        // output: cumulative mesh charges for guard cells
        double* cumweight_ij, double* cumweight_i1j,
        double* cumweight_ij1, double* cumweight_i1j1)
/**
    Calculate the Cloud-in-Cell weights for all particles within a
    guard cell.

    This node-based algorithm expects particle arrays sorted
    by their node id. For each node, the corresponding guard cell
    is spanned from the nodes spatial indices i, j to i+1, j+1,
    (Therefore, the guard cells at the rear boundary nodes do not
    get any contribution for the guard cell nodes that lie outside of
    the original mesh. E.g. i+1 == ny gets a zero entry for sure!)
    Within a guard cell, all particles are weighted according to
    their distance to the respective bounding node.

    The weights of each particle are summed up per guard cell node and
    written back to the global arrays cumweight_ij etc.

    The index arrays lower_bounds and upper_bounds
    indicate the start and end indices
    within the sorted particle arrays for each node id. The respective
    node id is identical to the index within lower_bounds and
    upper_bounds.
*/
{
    double l_cumweight_ij,  l_cumweight_i1j,  l_cumweight_ij1,  l_cumweight_i1j1;
    int i, j;
    double x0bydx = x0/dx;
    double y0bydy = y0/dy;
    double dx_rel, dy_rel;
    // grid-stride loop
    for (int nid = blockIdx.x * blockDim.x + threadIdx.x;
         nid < n_nodes;
         nid += blockDim.x * gridDim.x)
    {
        j = nid % nx; //& (nx-1); //
        i = ((nid - j) / nx); //& (ny-1); //

        l_cumweight_ij = 0.;   l_cumweight_i1j = 0.; l_cumweight_ij1 = 0.;
        l_cumweight_i1j1 = 0.;
        for (int pid = lower_bounds[nid]; pid < upper_bounds[nid]; pid++)
        {
            dx_rel = x_sorted[pid]/dx - x0bydx - j;
            dy_rel = y_sorted[pid]/dy - y0bydy - i;

            // locally calculate the weights for all 8 nodes of current guard cell
            l_cumweight_ij +=    (1-dx_rel)*(1-dy_rel);
            l_cumweight_i1j +=   (1-dx_rel)*(dy_rel)  ;
            l_cumweight_ij1 +=   (dx_rel)  *(1-dy_rel);
            l_cumweight_i1j1 +=  (dx_rel)  *(dy_rel)  ;
        }
        cumweight_ij[nid] =   l_cumweight_ij;   cumweight_i1j[nid] =   l_cumweight_i1j;
        cumweight_ij1[nid] =  l_cumweight_ij1;  cumweight_i1j1[nid] =  l_cumweight_i1j1;
    }
}

__global__ void join_guard_cells_2d(
        double* cumweight_ij, double* cumweight_i1j,
        double* cumweight_ij1, double* cumweight_i1j1,
        int n_nodes, int nx, int ny,
        double* mesh_charges)
/**

*/
{
    int i, j, ij, i1j, ij1, i1j1;
    // grid-stride loop
    for (int nid = blockIdx.x * blockDim.x + threadIdx.x;
         nid < n_nodes;
         nid += blockDim.x * gridDim.x)
    {
        j = nid % nx; //& (nx-1); //
        i = ((nid - j) / nx); //& (ny-1); //
        if (j == 0 || j == nx - 1 || i == 0 || i == ny - 1)
        {
            continue;
        }

        ij = nid;                     // nx*i     + j
        i1j = nid - nx;               // nx*(i-1) + j
        ij1 = nid - 1;                // nx*i     + j-1
        i1j1 = nid - nx - 1;          // nx*(i-1) + j-1


        mesh_charges[nid] =   cumweight_ij[ij]     + cumweight_i1j[i1j]
                            + cumweight_ij1[ij1]   + cumweight_i1j1[i1j1];

    }
}


} /* end extern C */
